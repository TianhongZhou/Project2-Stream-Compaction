#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void upSweepKern(int n, int* odata, int d) {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;

            if (k >= n || k % (1 << (d + 1)) != 0) {
                return;
            }

            odata[k + (1 << (d + 1)) - 1] += odata[k + (1 << d) - 1];
        }

        __global__ void downSweepKern(int n, int* odata, int d) {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;

            if (k >= n || k % (1 << (d + 1)) != 0) {
                return;
            }

            int t = odata[k + (1 << d) - 1];
            odata[k + (1 << d) - 1] = odata[k + (1 << (d + 1)) - 1];
            odata[k + (1 << (d + 1)) - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool time) {
            int logn = ilog2ceil(n);
            int new_n = 1 << logn;

            int* dev_buffer;

            hipMalloc((void**)&dev_buffer, new_n * sizeof(int));
            hipMemset(dev_buffer, 0, new_n * sizeof(int));
            hipMemcpy(dev_buffer, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 128;
            dim3 blockNum((new_n + blockSize - 1) / blockSize);

            if (time) timer().startGpuTimer();
            // TODO
            // Up-Sweep
            for (int d = 0; d <= logn - 1; d++) {
                upSweepKern<<<blockNum, blockSize>>>(new_n, dev_buffer, d);
                hipDeviceSynchronize();
            }

            hipMemset(&dev_buffer[new_n - 1], 0, sizeof(int));

            // Down-Sweep
            for (int d = logn - 1; d >= 0; d--) {
                downSweepKern<<<blockNum, blockSize>>>(new_n, dev_buffer, d);
                hipDeviceSynchronize();
            }

            if (time) timer().endGpuTimer();

            hipMemcpy(odata, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_buffer);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            int* dev_bool;
            int* dev_indicies;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMalloc((void**)&dev_bool, n * sizeof(int));
            hipMalloc((void**)&dev_indicies, n * sizeof(int));

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 128;
            dim3 blockNum((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            // TODO
            StreamCompaction::Common::kernMapToBoolean<<<blockNum, blockSize>>>(n, dev_bool, dev_idata);
            hipDeviceSynchronize();

            scan(n, dev_indicies, dev_bool, false);
            hipDeviceSynchronize();

            StreamCompaction::Common::kernScatter<<<blockNum, blockSize>>>(n, dev_odata, dev_idata, dev_bool, dev_indicies);
            hipDeviceSynchronize();

            timer().endGpuTimer();

            int lastBool;
            int lastIndex;

            hipMemcpy(&lastBool, dev_bool + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastIndex, dev_indicies + n - 1, sizeof(int), hipMemcpyDeviceToHost);

            int count = lastBool + lastIndex;
            hipMemcpy(odata, dev_odata, count * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata);
            hipFree(dev_idata);
            hipFree(dev_bool);
            hipFree(dev_indicies);

            return count;
        }
    }
}
